#include "hip/hip_runtime.h"
#include "bico_config.h"
#include "glu_kernels.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

void verify_results(const std::vector<half> &ref, const std::vector<half> &res,
                    const std::string &name) {
  bool correct = true;
  float max_err = 0.0f;
  for (size_t i = 0; i < ref.size(); ++i) {
    float r = __half2float(ref[i]);
    float s = __half2float(res[i]);
    float err = std::abs(r - s);
    if (err > 1e-2f) {
      correct = false;
      std::cerr << "Verification FAILED for " << name << " at index " << i
                << ". Ref: " << r << ", Res: " << s << " (err=" << err << ")"
                << std::endl;
      break;
    }
    max_err = std::max(max_err, err);
  }
  if (correct)
    std::cout << "Verification PASSED for " << name << " (max err: " << max_err
              << ")" << std::endl;
}

float run_kernel_with_params(int block_k, int num_warps, float smem_kb,
                             float occupancy, const half *d_A,
                             const half *d_W_up, const half *d_W_gate,
                             half *d_gated_result, int M, int N, int K) {
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  const size_t half_size = sizeof(half);
  const size_t static_smem_bytes =
      2 * TILE_ELEMENTS * sizeof(float) + TILE_ELEMENTS * half_size;

  int max_smem_elems = static_cast<int>(smem_kb * 1024 / half_size);
  int wmma_elems = block_k * WMMA_N;
  int available_elems = max_smem_elems - 2 * wmma_elems;
  int effective_block_k = block_k;

  if (occupancy < 0.5f)
    effective_block_k = std::min(block_k * 2, available_elems / WMMA_M);

  size_t dynamic_smem_bytes =
      (static_cast<size_t>(WMMA_M) * effective_block_k +
       2 * static_cast<size_t>(effective_block_k) * WMMA_N) *
      half_size;
  size_t total_smem_bytes = dynamic_smem_bytes + static_smem_bytes;

  int dev_id;
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDevice(&dev_id));
  CUDA_CHECK(hipGetDeviceProperties(&prop, dev_id));

  if (total_smem_bytes > static_cast<size_t>(prop.sharedMemPerBlock)) {
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    return std::numeric_limits<float>::max();
  }

  dim3 grid_fused((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
  dim3 block_size(32 * num_warps);
  float latency_ms = 0;

  try {
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(start));

    if (block_k == 64 && num_warps == 2) {
      glu_fused_up_gate_swiglu_budget<64, 2>
          <<<grid_fused, block_size, dynamic_smem_bytes>>>(
              d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
              static_cast<int>(smem_kb), occupancy);
    } else if (block_k == 128 && num_warps == 4) {
      glu_fused_up_gate_swiglu_budget<128, 4>
          <<<grid_fused, block_size, dynamic_smem_bytes>>>(
              d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
              static_cast<int>(smem_kb), occupancy);
    } else {
      glu_fused_up_gate_swiglu_budget<64, 4>
          <<<grid_fused, block_size, dynamic_smem_bytes>>>(
              d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
              static_cast<int>(smem_kb), occupancy);
    }

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&latency_ms, start, stop));
  } catch (...) {
    latency_ms = std::numeric_limits<float>::max();
  }

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  return latency_ms;
}

Evaluator createEvaluator(const half *d_A, const half *d_W_up,
                          const half *d_W_gate, half *d_gated_result, int M,
                          int N, int K, half *d_W_down, half *d_output) {
  return [=](const SearchConfig &config) -> KernelResult {
    auto search_start = std::chrono::high_resolution_clock::now();
    float best_latency = std::numeric_limits<float>::max();

    std::mt19937 rng(std::random_device{}());
    std::uniform_int_distribution<int> block_k_dist(32, 128);
    std::uniform_int_distribution<int> warps_dist(2, 8);
    std::uniform_real_distribution<float> smem_dist(32.0f, 128.0f);
    std::uniform_real_distribution<float> occ_dist(0.1f, 0.9f);

    int evaluations = 0;

    while (true) {
      auto current_time = std::chrono::high_resolution_clock::now();
      double elapsed =
          std::chrono::duration<double>(current_time - search_start).count();
      if (elapsed >= config.search_time_budget)
        break;

      int block_k = block_k_dist(rng);
      int num_warps = warps_dist(rng);
      float smem_kb = smem_dist(rng);
      float occupancy = occ_dist(rng);

      float latency =
          run_kernel_with_params(block_k, num_warps, smem_kb, occupancy, d_A,
                                 d_W_up, d_W_gate, d_gated_result, M, N, K);

      if (latency < best_latency)
        best_latency = latency;

      evaluations++;
      dim3 grid_down((K + WMMA_K - 1) / WMMA_K, (M + WMMA_M - 1) / WMMA_M);
      glu_kernel3_down_gemm<<<grid_down, dim3(32)>>>(d_gated_result, d_W_down,
                                                     d_output, M, N, K);
    }

    std::cout << "Evaluated " << evaluations << " configurations in "
              << config.search_time_budget << " seconds" << std::endl;

    return KernelResult(config, best_latency,
                        best_latency < std::numeric_limits<float>::max());
  };
}

int main() {
  const int M = 1024;
  const int K_hidden = 4096;
  const int N_inter = 12288;
  std::cout << "Problem Size: A(" << M << "x" << K_hidden << ") @ W("
            << K_hidden << "x" << N_inter << ") -> down (" << N_inter << "x"
            << K_hidden << ")" << std::endl;
  std::vector<half> h_A(M * K_hidden);
  std::vector<half> h_W_up(K_hidden * N_inter);
  std::vector<half> h_W_gate(K_hidden * N_inter);
  std::vector<half> h_W_down(N_inter * K_hidden);
  std::vector<half> h_output_baseline(M * K_hidden);
  std::vector<half> h_output_fused(M * K_hidden);
  std::mt19937 rng(1337);
  std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
  for (auto &val : h_A)
    val = __float2half(dist(rng));
  for (auto &val : h_W_up)
    val = __float2half(dist(rng));
  for (auto &val : h_W_gate)
    val = __float2half(dist(rng));
  for (auto &val : h_W_down)
    val = __float2half(dist(rng));
  half *d_A, *d_W_up, *d_W_gate, *d_W_down, *d_output;
  half *d_up_proj, *d_gate_proj, *d_gated_result;
  CUDA_CHECK(hipMalloc(&d_A, M * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_up, K_hidden * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_gate, K_hidden * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_down, N_inter * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_output, M * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_up_proj, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_gate_proj, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_gated_result, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), M * K_hidden * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_up, h_W_up.data(),
                        K_hidden * N_inter * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_gate, h_W_gate.data(),
                        K_hidden * N_inter * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_down, h_W_down.data(),
                        N_inter * K_hidden * sizeof(half),
                        hipMemcpyHostToDevice));
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  float ms_baseline = 0;
  dim3 warp_block(32, 1);
  dim3 grid_up((N_inter + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
  dim3 grid_elem((M * N_inter + 255) / 256);
  dim3 block_elem(256);
  dim3 grid_down((K_hidden + WMMA_K - 1) / WMMA_K, (M + WMMA_M - 1) / WMMA_M);
  std::cout << "\n--- Running Baseline (3 Kernels) ---" << std::endl;
  CUDA_CHECK(hipEventRecord(start));
  glu_kernel1_up_gate_gemm<<<grid_up, warp_block>>>(
      d_A, d_W_up, d_W_gate, d_up_proj, d_gate_proj, M, N_inter, K_hidden);
  glu_kernel2_elementwise_swiglu<<<grid_elem, block_elem>>>(
      d_up_proj, d_gate_proj, d_gated_result, M, N_inter);
  glu_kernel3_down_gemm<<<grid_down, warp_block>>>(
      d_gated_result, d_W_down, d_output, M, N_inter, K_hidden);
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&ms_baseline, start, stop));
  std::cout << "Baseline Latency: " << ms_baseline << " ms" << std::endl;
  CUDA_CHECK(hipMemcpy(h_output_baseline.data(), d_output,
                        M * K_hidden * sizeof(half), hipMemcpyDeviceToHost));
  auto evaluator = createEvaluator(d_A, d_W_up, d_W_gate, d_gated_result, M,
                                   N_inter, K_hidden, d_W_down, d_output);
  BICOExplorer explorer(evaluator);
  std::cout << "\n--- Exploring with Time Budget ---" << std::endl;
  explorer.exploreWithTimeBudget(10.0);
  KernelResult best_result = explorer.findBestConfig();
  std::cout << "Best configuration: " << best_result.config
            << " with latency: " << best_result.latency_ms << " ms"
            << std::endl;
  std::cout << "\n--- Final Run with Best Configuration ---" << std::endl;
  KernelResult final_result = evaluator(SearchConfig(1.0));
  if (final_result.valid) {
    std::cout << "Final latency: " << final_result.latency_ms << " ms"
              << std::endl;
    CUDA_CHECK(hipMemcpy(h_output_fused.data(), d_output,
                          M * K_hidden * sizeof(half), hipMemcpyDeviceToHost));
    std::cout << "\n--- Verification ---" << std::endl;
    verify_results(h_output_baseline, h_output_fused, "Optimized vs. Baseline");
    std::cout << "\n--- Performance Summary ---" << std::endl;
    if (final_result.latency_ms > 0)
      std::cout << "Speedup from Optimization: "
                << (ms_baseline / final_result.latency_ms) << "x" << std::endl;
  } else {
    std::cout << "Best configuration is invalid!" << std::endl;
  }
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_W_up));
  CUDA_CHECK(hipFree(d_W_gate));
  CUDA_CHECK(hipFree(d_W_down));
  CUDA_CHECK(hipFree(d_output));
  CUDA_CHECK(hipFree(d_up_proj));
  CUDA_CHECK(hipFree(d_gate_proj));
  CUDA_CHECK(hipFree(d_gated_result));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  return 0;
}
