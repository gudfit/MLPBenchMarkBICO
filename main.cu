#include "hip/hip_runtime.h"
#include "bico_config.h"
#include "glu_kernels.h"
#include <algorithm>
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

void verify_results(const std::vector<half> &ref, const std::vector<half> &res,
                    const std::string &name) {
  bool correct = true;
  float max_err = 0.0f;
  for (size_t i = 0; i < ref.size(); ++i) {
    float r = __half2float(ref[i]);
    float s = __half2float(res[i]);
    float err = std::abs(r - s);
    if (err > 1e-2f) {
      correct = false;
      std::cerr << "Verification FAILED for " << name << " at index " << i
                << ". Ref: " << r << ", Res: " << s << " (err=" << err << ")"
                << std::endl;
      break;
    }
    max_err = std::max(max_err, err);
  }
  if (correct)
    std::cout << "Verification PASSED for " << name << " (max err: " << max_err
              << ")" << std::endl;
}

Evaluator createEvaluator(const half *d_A, const half *d_W_up,
                          const half *d_W_gate, half *d_gated_result, int M,
                          int N, int K, half *d_W_down, half *d_output) {
  return [=](const BudgetConfig &config) -> KernelResult {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    int block_k = 64;
    int num_warps = 4;

    if (config.smem_budget < 48) {
      block_k = 32;
      num_warps = 2;
    } else if (config.smem_budget > 96) {
      block_k = 128;
      num_warps = 8;
    }

    size_t smem_size =
        (WMMA_M * block_k + 2 * block_k * WMMA_N) * sizeof(half) +
        2 * TILE_ELEMENTS * sizeof(float) + TILE_ELEMENTS * sizeof(half);

    int dev_id;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&dev_id));
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev_id));

    if (smem_size > prop.sharedMemPerBlock) {
      CUDA_CHECK(hipEventDestroy(start));
      CUDA_CHECK(hipEventDestroy(stop));
      return KernelResult(config, 0, false);
    }

    dim3 grid_fused((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block_size(32 * num_warps);

    float latency_ms = 0;

    try {
      if (block_k == 32 && num_warps == 2) {
        glu_fused_up_gate_swiglu_budget<32, 2>
            <<<grid_fused, block_size, smem_size>>>(
                d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                config.smem_budget, config.occ_budget);
      } else if (block_k == 128 && num_warps == 8) {
        glu_fused_up_gate_swiglu_budget<128, 8>
            <<<grid_fused, block_size, smem_size>>>(
                d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                config.smem_budget, config.occ_budget);
      } else {
        glu_fused_up_gate_swiglu_budget<64, 4>
            <<<grid_fused, block_size, smem_size>>>(
                d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                config.smem_budget, config.occ_budget);
      }

      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipEventRecord(start));
      for (int i = 0; i < 10; ++i) {
        if (block_k == 32 && num_warps == 2) {
          glu_fused_up_gate_swiglu_budget<32, 2>
              <<<grid_fused, block_size, smem_size>>>(
                  d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                  config.smem_budget, config.occ_budget);
        } else if (block_k == 128 && num_warps == 8) {
          glu_fused_up_gate_swiglu_budget<128, 8>
              <<<grid_fused, block_size, smem_size>>>(
                  d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                  config.smem_budget, config.occ_budget);
        } else {
          glu_fused_up_gate_swiglu_budget<64, 4>
              <<<grid_fused, block_size, smem_size>>>(
                  d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                  config.smem_budget, config.occ_budget);
        }
      }
      CUDA_CHECK(hipEventRecord(stop));
      CUDA_CHECK(hipEventSynchronize(stop));
      CUDA_CHECK(hipEventElapsedTime(&latency_ms, start, stop));
      latency_ms /= 10.0f;
      dim3 grid_down((K + WMMA_K - 1) / WMMA_K, (M + WMMA_M - 1) / WMMA_M);
      glu_kernel3_down_gemm<<<grid_down, dim3(32)>>>(d_gated_result, d_W_down,
                                                     d_output, M, N, K);

      CUDA_CHECK(hipDeviceSynchronize());
    } catch (...) {
      CUDA_CHECK(hipEventDestroy(start));
      CUDA_CHECK(hipEventDestroy(stop));
      return KernelResult(config, 0, false);
    }

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    return KernelResult(config, latency_ms, true);
  };
}

int main() {
  const int M = 1024;
  const int K_hidden = 4096;
  const int N_inter = 12288;

  std::cout << "Problem Size: A(" << M << "x" << K_hidden << ") @ W("
            << K_hidden << "x" << N_inter << ") -> down (" << N_inter << "x"
            << K_hidden << ")" << std::endl;

  std::vector<half> h_A(M * K_hidden);
  std::vector<half> h_W_up(K_hidden * N_inter);
  std::vector<half> h_W_gate(K_hidden * N_inter);
  std::vector<half> h_W_down(N_inter * K_hidden);
  std::vector<half> h_output_baseline(M * K_hidden);
  std::vector<half> h_output_fused(M * K_hidden);
  std::mt19937 rng(1337);
  std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
  for (auto &val : h_A)
    val = __float2half(dist(rng));
  for (auto &val : h_W_up)
    val = __float2half(dist(rng));
  for (auto &val : h_W_gate)
    val = __float2half(dist(rng));
  for (auto &val : h_W_down)
    val = __float2half(dist(rng));
  half *d_A, *d_W_up, *d_W_gate, *d_W_down, *d_output;
  half *d_up_proj, *d_gate_proj, *d_gated_result;
  CUDA_CHECK(hipMalloc(&d_A, M * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_up, K_hidden * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_gate, K_hidden * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_down, N_inter * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_output, M * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_up_proj, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_gate_proj, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_gated_result, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), M * K_hidden * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_up, h_W_up.data(),
                        K_hidden * N_inter * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_gate, h_W_gate.data(),
                        K_hidden * N_inter * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_down, h_W_down.data(),
                        N_inter * K_hidden * sizeof(half),
                        hipMemcpyHostToDevice));
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  float ms_baseline = 0, ms_fused = 0;

  dim3 warp_block(32, 1);
  dim3 grid_up((N_inter + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
  dim3 grid_elem((M * N_inter + 255) / 256);
  dim3 block_elem(256);
  dim3 grid_down((K_hidden + WMMA_K - 1) / WMMA_K, (M + WMMA_M - 1) / WMMA_M);

  std::cout << "\n--- Running Baseline (3 Kernels) ---" << std::endl;
  CUDA_CHECK(hipEventRecord(start));
  glu_kernel1_up_gate_gemm<<<grid_up, warp_block>>>(
      d_A, d_W_up, d_W_gate, d_up_proj, d_gate_proj, M, N_inter, K_hidden);
  glu_kernel2_elementwise_swiglu<<<grid_elem, block_elem>>>(
      d_up_proj, d_gate_proj, d_gated_result, M, N_inter);
  glu_kernel3_down_gemm<<<grid_down, warp_block>>>(
      d_gated_result, d_W_down, d_output, M, N_inter, K_hidden);
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&ms_baseline, start, stop));
  std::cout << "Baseline Latency: " << ms_baseline << " ms" << std::endl;
  CUDA_CHECK(hipMemcpy(h_output_baseline.data(), d_output,
                        M * K_hidden * sizeof(half), hipMemcpyDeviceToHost));

  auto evaluator = createEvaluator(d_A, d_W_up, d_W_gate, d_gated_result, M,
                                   N_inter, K_hidden, d_W_down, d_output);
  BICOExplorer explorer(evaluator);
  std::cout << "\n--- Exploring Budget Frontier ---" << std::endl;
  explorer.exploreFrontier();
  BudgetConfig best_config = explorer.findBestConfig();
  std::cout << "Best configuration: " << best_config << std::endl;
  std::cout << "\n--- Final Run with Best Configuration ---" << std::endl;
  KernelResult final_result = evaluator(best_config);

  if (final_result.valid) {
    std::cout << "Final latency: " << final_result.latency_ms << " ms"
              << std::endl;
    CUDA_CHECK(hipMemcpy(h_output_fused.data(), d_output,
                          M * K_hidden * sizeof(half), hipMemcpyDeviceToHost));

    std::cout << "\n--- Verification ---" << std::endl;
    verify_results(h_output_baseline, h_output_fused, "Optimized vs. Baseline");
    std::cout << "\n--- Performance Summary ---" << std::endl;
    if (final_result.latency_ms > 0)
      std::cout << "Speedup from Optimization: "
                << (ms_baseline / final_result.latency_ms) << "x" << std::endl;
  } else {
    std::cout << "Best configuration is invalid!" << std::endl;
  }

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_W_up));
  CUDA_CHECK(hipFree(d_W_gate));
  CUDA_CHECK(hipFree(d_W_down));
  CUDA_CHECK(hipFree(d_output));
  CUDA_CHECK(hipFree(d_up_proj));
  CUDA_CHECK(hipFree(d_gate_proj));
  CUDA_CHECK(hipFree(d_gated_result));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}
