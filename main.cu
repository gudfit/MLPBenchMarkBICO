#include "hip/hip_runtime.h"
#include "bico_config.h"
#include "glu_kernels.h"
#include <algorithm>
#include <cmath>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

// Helper function to verify results between reference and test implementations
void verify_results(const std::vector<half> &ref, const std::vector<half> &res,
                    const std::string &name) {
  bool correct = true;
  float max_err = 0.0f;
  for (size_t i = 0; i < ref.size(); ++i) {
    float r = __half2float(ref[i]);
    float s = __half2float(res[i]);
    float err = std::abs(r - s);
    if (err > 1e-2f) {
      correct = false;
      std::cerr << "Verification FAILED for " << name << " at index " << i
                << ". Ref: " << r << ", Res: " << s << " (err=" << err << ")"
                << std::endl;
      break;
    }
    max_err = std::max(max_err, err);
  }
  if (correct)
    std::cout << "Verification PASSED for " << name << " (max err: " << max_err
              << ")" << std::endl;
}

Evaluator createEvaluator(const half *d_A, const half *d_W_up,
                          const half *d_W_gate, half *d_gated_result, int M,
                          int N, int K, half *d_W_down, half *d_output) {
  constexpr size_t half_size = sizeof(half);
  constexpr size_t static_smem_bytes =
      2 * TILE_ELEMENTS * sizeof(float) + TILE_ELEMENTS * half_size;
  return [=](const BudgetConfig &config) -> KernelResult {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    int block_k = 64;
    int num_warps = 4;
    float smem_kb = config.smem_budget;
    int max_smem_elems = static_cast<int>(smem_kb * 1024 / half_size);
    int wmma_elems = block_k * WMMA_N;
    int available_elems = max_smem_elems - 2 * wmma_elems;
    int effective_block_k = block_k;
    if (config.occ_budget < 0.5f)
      effective_block_k = std::min(block_k * 2, available_elems / WMMA_M);

    if (smem_kb <= 48 && config.occ_budget <= 0.4f) {
      block_k = 64;
      num_warps = 2;
    } else if (smem_kb > 96 && config.occ_budget <= 0.3f) {
      block_k = 128;
      num_warps = 4;
    } else {
      block_k = 64;
      num_warps = 4;
    }
    wmma_elems = block_k * WMMA_N;
    available_elems = max_smem_elems - 2 * wmma_elems;
    effective_block_k = block_k;
    if (config.occ_budget < 0.5f)
      effective_block_k =
          std::min(block_k * 2, std::max(0, available_elems) / WMMA_M);

    size_t dynamic_smem_bytes =
        (static_cast<size_t>(WMMA_M) * effective_block_k +
         2 * static_cast<size_t>(effective_block_k) * WMMA_N) *
        half_size;
    size_t total_smem_bytes = dynamic_smem_bytes + static_smem_bytes;

    int dev_id;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&dev_id));
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev_id));
    if (total_smem_bytes > static_cast<size_t>(prop.sharedMemPerBlock)) {
      CUDA_CHECK(hipEventDestroy(start));
      CUDA_CHECK(hipEventDestroy(stop));
      return KernelResult(config, 0, false);
    }

    dim3 grid_fused((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block_size(32 * num_warps);
    float latency_ms = 0;

    try {
      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipEventRecord(start));
      for (int i = 0; i < 5; ++i) {
        if (smem_kb <= 48 && config.occ_budget <= 0.4f) {
          glu_fused_up_gate_swiglu_budget<64, 2>
              <<<grid_fused, block_size, dynamic_smem_bytes>>>(
                  d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                  static_cast<int>(smem_kb), config.occ_budget);
        } else if (smem_kb > 96 && config.occ_budget <= 0.3f) {
          glu_fused_up_gate_swiglu_budget<128, 4>
              <<<grid_fused, block_size, dynamic_smem_bytes>>>(
                  d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                  static_cast<int>(smem_kb), config.occ_budget);
        } else {
          glu_fused_up_gate_swiglu_budget<64, 4>
              <<<grid_fused, block_size, dynamic_smem_bytes>>>(
                  d_A, d_W_up, d_W_gate, d_gated_result, M, N, K,
                  static_cast<int>(smem_kb), config.occ_budget);
        }
      }
      CUDA_CHECK(hipEventRecord(stop));
      CUDA_CHECK(hipEventSynchronize(stop));
      CUDA_CHECK(hipEventElapsedTime(&latency_ms, start, stop));
      latency_ms /= 5.0f;

      dim3 grid_down((K + WMMA_K - 1) / WMMA_K, (M + WMMA_M - 1) / WMMA_M);
      glu_kernel3_down_gemm<<<grid_down, dim3(32)>>>(d_gated_result, d_W_down,
                                                     d_output, M, N, K);

      CUDA_CHECK(hipDeviceSynchronize());
    } catch (...) {
      CUDA_CHECK(hipEventDestroy(start));
      CUDA_CHECK(hipEventDestroy(stop));
      return KernelResult(config, 0, false);
    }

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    return KernelResult(config, latency_ms, true);
  };
}

int main() {
  const int M = 1024;
  const int K_hidden = 4096;
  const int N_inter = 12288;
  std::cout << "Problem Size: A(" << M << "x" << K_hidden << ") @ W("
            << K_hidden << "x" << N_inter << ") -> down (" << N_inter << "x"
            << K_hidden << ")" << std::endl;
  std::vector<half> h_A(M * K_hidden);
  std::vector<half> h_W_up(K_hidden * N_inter);
  std::vector<half> h_W_gate(K_hidden * N_inter);
  std::vector<half> h_W_down(N_inter * K_hidden);
  std::vector<half> h_output_baseline(M * K_hidden);
  std::vector<half> h_output_fused(M * K_hidden);
  std::mt19937 rng(1337);
  std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
  for (auto &val : h_A)
    val = __float2half(dist(rng));
  for (auto &val : h_W_up)
    val = __float2half(dist(rng));
  for (auto &val : h_W_gate)
    val = __float2half(dist(rng));
  for (auto &val : h_W_down)
    val = __float2half(dist(rng));
  half *d_A, *d_W_up, *d_W_gate, *d_W_down, *d_output;
  half *d_up_proj, *d_gate_proj, *d_gated_result;
  CUDA_CHECK(hipMalloc(&d_A, M * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_up, K_hidden * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_gate, K_hidden * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_W_down, N_inter * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_output, M * K_hidden * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_up_proj, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_gate_proj, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMalloc(&d_gated_result, M * N_inter * sizeof(half)));
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), M * K_hidden * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_up, h_W_up.data(),
                        K_hidden * N_inter * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_gate, h_W_gate.data(),
                        K_hidden * N_inter * sizeof(half),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W_down, h_W_down.data(),
                        N_inter * K_hidden * sizeof(half),
                        hipMemcpyHostToDevice));
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  float ms_baseline = 0;
  dim3 warp_block(32, 1);
  dim3 grid_up((N_inter + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
  dim3 grid_elem((M * N_inter + 255) / 256);
  dim3 block_elem(256);
  dim3 grid_down((K_hidden + WMMA_K - 1) / WMMA_K, (M + WMMA_M - 1) / WMMA_M);
  std::cout << "\n--- Running Baseline (3 Kernels) ---" << std::endl;
  CUDA_CHECK(hipEventRecord(start));
  glu_kernel1_up_gate_gemm<<<grid_up, warp_block>>>(
      d_A, d_W_up, d_W_gate, d_up_proj, d_gate_proj, M, N_inter, K_hidden);
  glu_kernel2_elementwise_swiglu<<<grid_elem, block_elem>>>(
      d_up_proj, d_gate_proj, d_gated_result, M, N_inter);
  glu_kernel3_down_gemm<<<grid_down, warp_block>>>(
      d_gated_result, d_W_down, d_output, M, N_inter, K_hidden);
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&ms_baseline, start, stop));
  std::cout << "Baseline Latency: " << ms_baseline << " ms" << std::endl;
  CUDA_CHECK(hipMemcpy(h_output_baseline.data(), d_output,
                        M * K_hidden * sizeof(half), hipMemcpyDeviceToHost));
  auto evaluator = createEvaluator(d_A, d_W_up, d_W_gate, d_gated_result, M,
                                   N_inter, K_hidden, d_W_down, d_output);
  BICOExplorer explorer(evaluator);
  std::cout << "\n--- Exploring Budget Frontier ---" << std::endl;
  explorer.exploreFrontier();
  BudgetConfig best_config = explorer.findBestConfig();
  std::cout << "Best configuration: " << best_config << std::endl;
  std::cout << "\n--- Final Run with Best Configuration ---" << std::endl;
  KernelResult final_result = evaluator(best_config);

  if (final_result.valid) {
    std::cout << "Final latency: " << final_result.latency_ms << " ms"
              << std::endl;
    CUDA_CHECK(hipMemcpy(h_output_fused.data(), d_output,
                          M * K_hidden * sizeof(half), hipMemcpyDeviceToHost));

    std::cout << "\n--- Verification ---" << std::endl;
    verify_results(h_output_baseline, h_output_fused, "Optimized vs. Baseline");
    std::cout << "\n--- Performance Summary ---" << std::endl;
    if (final_result.latency_ms > 0)
      std::cout << "Speedup from Optimization: "
                << (ms_baseline / final_result.latency_ms) << "x" << std::endl;
  } else {
    std::cout << "Best configuration is invalid!" << std::endl;
  }
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_W_up));
  CUDA_CHECK(hipFree(d_W_gate));
  CUDA_CHECK(hipFree(d_W_down));
  CUDA_CHECK(hipFree(d_output));
  CUDA_CHECK(hipFree(d_up_proj));
  CUDA_CHECK(hipFree(d_gate_proj));
  CUDA_CHECK(hipFree(d_gated_result));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}
